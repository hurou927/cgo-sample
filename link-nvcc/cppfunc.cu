
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define NUMTHREADS 32

__global__ void kernel(){
	int tx = blockDim.x*blockIdx.x+threadIdx.x;
    printf("%d\n",tx);
}



extern "C" {

int cppfunc(int a){
    int numthreads= NUMTHREADS;
    int numblocks = 2;
    printf("cppfunc:%d blocks %d\n", a, numblocks);
    std::cout<<"Hello"<<std::endl;

    std::cout << "kernel:" << a << std::endl;
    hipDeviceSynchronize();
    kernel <<< numblocks , numthreads >>> ();
    hipDeviceSynchronize();

    return 927;
}

}
